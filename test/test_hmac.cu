#include "hip/hip_runtime.h"
//  Copyright Matt Borland 2024
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <hip/hip_runtime.h>
#include <boost/crypt2/mac/hmac.hpp>
#include <boost/crypt2/hash/sha3_512.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"
#include "generate_random_strings.hpp"
#include <iostream>
#include <iomanip>
#include <exception>
#include <memory>
#include <span>

using digest_type = typename boost::crypt::sha3_512_hasher::return_type;

// The kernel function
__global__ void cuda_test(char** in, digest_type* out, int numElements)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numElements)
    {
        boost::crypt::hmac<boost::crypt::sha3_512_hasher> hmac_class;
        cuda::std::span<char> in_span {in[i], static_cast<cuda::std::size_t>(64)};
        hmac_class.init(in_span);
        hmac_class.process_bytes(in_span);
        hmac_class.finalize();
        out[i] = hmac_class.get_digest().value();
    }
}

int main()
{
    try
    {
        // Error code to check return values for CUDA calls
        hipError_t err = hipSuccess;

        // Print the vector length to be used, and compute its size
        constexpr int numElements = 50000;
        constexpr std::size_t elementSize = 64;

        std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

        // Allocate the managed input vector A
        char** input_vector1;
        hipMallocManaged(&input_vector1, numElements * sizeof(char*));

        // Allocate the managed output vector C
        cuda_managed_ptr<digest_type> output_vector(numElements);

        for (int i = 0; i < numElements; ++i)
        {
            hipMallocManaged(&input_vector1[i], elementSize * sizeof(char));
            if (input_vector1[i] == nullptr)
            {
                throw std::runtime_error("Failed to allocate memory for input_vector1");
            }
            boost::crypt::generate_random_string(input_vector1[i], elementSize);
        }

        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
        std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

        watch w;
        cuda_test<<<blocksPerGrid, threadsPerBlock>>>(input_vector1, output_vector.get(), numElements);
        hipDeviceSynchronize();
        std::cout << "CUDA kernal done in " << w.elapsed() << "s" << std::endl;

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
            return EXIT_FAILURE;
        }

        // Verify that the result vector is correct
        std::vector<digest_type> results;
        results.reserve(numElements);
        w.reset();
        for(int i = 0; i < numElements; ++i)
        {
            std::span<char> in(input_vector1[i], elementSize);
            boost::crypt::hmac<boost::crypt::sha3_512_hasher> hmac_class;
            auto in_span {cuda::std::span(in, 64)};
            hmac_class.init(in_span);
            hmac_class.process_bytes(in_span);
            hmac_class.finalize();
            results.emplace_back(hmac_class.get_digest().value());
        }
        double t = w.elapsed();

        // check the results
        for(int i = 0; i < numElements; ++i)
        {
            if (output_vector[i][0] != results[i][0])
            {
                std::cerr << "Result verification failed at element " << i << "!" << std::endl;
                return EXIT_FAILURE;
            }
        }

        std::cout << "Test PASSED with calculation time: " << t << "s" << std::endl;
        std::cout << "Done\n";

        // Cleanup all the memory we allocated
        for (int i = 0; i < numElements; ++i)
        {
            hipFree(input_vector1[i]);
        }
        hipFree(input_vector1);
    }
    catch (const std::exception& e)
    {
        std::cerr << "Terminated with exception: " << e.what() << std::endl;
    }
}
