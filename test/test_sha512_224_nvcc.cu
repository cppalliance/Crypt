#include "hip/hip_runtime.h"
//  Copyright Matt Borland 2024
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <boost/crypt/hash/sha512_224.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"
#include "generate_random_strings.hpp"
#include <iostream>
#include <iomanip>
#include <exception>
#include <memory>

#include <hip/hip_runtime.h>

using digest_type = boost::crypt::sha512_224_hasher::return_type;

// The kernel function
__global__ void cuda_test(char** in, digest_type* out, int numElements)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numElements)
    {
        out[i] = boost::crypt::sha512_224(in[i]);
    }
}

int main()
{
    try
    {
        // Error code to check return values for CUDA calls
        hipError_t err = hipSuccess;

        // Print the vector length to be used, and compute its size
        constexpr int numElements = 50000;
        constexpr std::size_t elementSize = 64;

        std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

        // Allocate the managed input vector A
        char** input_vector1;
        hipMallocManaged(&input_vector1, numElements * sizeof(char*));

        // Allocate the managed output vector C
        cuda_managed_ptr<digest_type> output_vector(numElements);

        for (int i = 0; i < numElements; ++i)
        {
            hipMallocManaged(&input_vector1[i], elementSize * sizeof(char));
            if (input_vector1[i] == nullptr)
            {
                throw std::runtime_error("Failed to allocate memory for input_vector1");
            }
            boost::crypt::generate_random_string(input_vector1[i], elementSize);
        }

        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
        std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

        watch w;
        cuda_test<<<blocksPerGrid, threadsPerBlock>>>(input_vector1, output_vector.get(), numElements);
        hipDeviceSynchronize();
        std::cout << "CUDA kernal done in " << w.elapsed() << "s" << std::endl;

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
            return EXIT_FAILURE;
        }

        // Verify that the result vector is correct
        std::vector<digest_type> results;
        results.reserve(numElements);
        w.reset();
        for(int i = 0; i < numElements; ++i)
        {
           results.emplace_back(boost::crypt::sha512_224(input_vector1[i]));
        }
        double t = w.elapsed();

        // check the results
        for(int i = 0; i < numElements; ++i)
        {
            if (output_vector[i][0] != results[i][0])
            {
                std::cerr << "Result verification failed at element " << i << "!" << std::endl;
                return EXIT_FAILURE;
            }
        }

        std::cout << "Test PASSED with calculation time: " << t << "s" << std::endl;
        std::cout << "Done\n";

        // Cleanup all the memory we allocated
        for (int i = 0; i < numElements; ++i)
        {
            hipFree(input_vector1[i]);
        }
        hipFree(input_vector1);
    }
    catch (const std::exception& e)
    {
        std::cerr << "Terminated with exception: " << e.what() << std::endl;
    }
}
